#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "../utils/SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

const int W = 40;
const int H = 12;

__global__ void Draw(char *frame) 
{
	// TODO: draw more complex things here
	// Do not just submit the original file provided by the TA!
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (y < H and x < W) 
    {
		char c;
		if (x == W-1) 
        {
			c = y == H-1 ? '\0' : '\n';
		}
        else if (y == 0 or y == H-1 or x == 0 or x == W-2) 
        {
			c = ':';
		}
        else if(
            y == 3 && x == 6 ||
            y == 3 && x == 7 ||
            y == 3 && x == 8 ||
            y == 3 && x == 9 ||
            y == 3 && x == 10 ||
            y == 3 && x == 28 ||
            y == 3 && x == 29 ||
            y == 3 && x == 30 ||
            y == 3 && x == 31 ||
            y == 3 && x == 32 ||
            y == 4 && x == 4 ||
            y == 4 && x == 5 ||
            y == 4 && x == 11 ||
            y == 4 && x == 12 ||
            y == 4 && x == 26 ||
            y == 4 && x == 27 ||
            y == 4 && x == 33 ||
            y == 4 && x == 34 ||
            y == 5 && x == 3 ||
            y == 5 && x == 4 ||
            y == 5 && x == 12 ||
            y == 5 && x == 13 ||
            y == 5 && x == 25 ||
            y == 5 && x == 26 ||
            y == 5 && x == 34 ||
            y == 5 && x == 35 ||
            y == 6 && x == 3 ||
            y == 6 && x == 4 ||
            y == 6 && x == 12 ||
            y == 6 && x == 13 ||
            y == 6 && x == 25 ||
            y == 6 && x == 26 ||
            y == 6 && x == 34 ||
            y == 6 && x == 35 ||
            y == 7 && x == 3 ||
            y == 7 && x == 4 ||
            y == 7 && x == 12 ||
            y == 7 && x == 13 ||
            y == 7 && x == 25 ||
            y == 7 && x == 26 ||
            y == 7 && x == 34 ||
            y == 7 && x == 35 ||
            y == 8 && x == 4 ||
            y == 8 && x == 5 ||
            y == 8 && x == 11 ||
            y == 8 && x == 12 ||
            y == 8 && x == 26 ||
            y == 8 && x == 27 ||
            y == 8 && x == 33 ||
            y == 8 && x == 34 ||
            y == 9 && x == 6 ||
            y == 9 && x == 7 ||
            y == 9 && x == 8 ||
            y == 9 && x == 9 ||
            y == 9 && x == 10 ||
            y == 9 && x == 28 ||
            y == 9 && x == 29 ||
            y == 9 && x == 30 ||
            y == 9 && x == 31 ||
            y == 9 && x == 32)
        {
            c = 'O';
        }
        else if(
            y == 3 && x == 19 ||
            y == 4 && x == 18 ||
            y == 4 && x == 19 ||
            y == 4 && x == 20 ||
            y == 5 && x == 17 ||
            y == 5 && x == 18 ||
            y == 5 && x == 20 ||
            y == 5 && x == 21 ||
            y == 6 && x == 16 ||
            y == 6 && x == 17 ||
            y == 6 && x == 21 ||
            y == 6 && x == 22 ||
            y == 7 && x == 15 ||
            y == 7 && x == 16 ||
            y == 7 && x == 17 ||
            y == 7 && x == 18 ||
            y == 7 && x == 19 ||
            y == 7 && x == 20 ||
            y == 7 && x == 21 ||
            y == 7 && x == 22 ||
            y == 7 && x == 23 ||
            y == 8 && x == 14 ||
            y == 8 && x == 15 ||
            y == 8 && x == 23 ||
            y == 8 && x == 24 ||
            y == 9 && x == 13 ||
            y == 9 && x == 14 ||
            y == 9 && x == 24 ||
            y == 9 && x == 25)
        {
            c = 'A';
        }
        else
        {
			c = ' ';
		}
		frame[y*W+x] = c;
	}
}

int main(int argc, char **argv)
{
	MemoryBuffer<char> frame(W*H);
	auto frame_smem = frame.CreateSync(W*H);
	CHECK;

	Draw<<<dim3((W-1)/16+1,(H-1)/12+1), dim3(16,12)>>>(frame_smem.get_gpu_wo());
	CHECK;

	puts(frame_smem.get_cpu_ro());
	CHECK;
	return 0;
}
